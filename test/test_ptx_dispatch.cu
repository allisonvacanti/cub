/******************************************************************************
 * Copyright (c) 2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <cub/util_ptx_dispatch.cuh>

#include "test_util.h"

struct no_type {};

using TestPolicy350 = hipcub::detail::ptx_tag<350, no_type>;
using TestPolicy600 = hipcub::detail::ptx_tag<600, no_type>;
using TestPolicy800 = hipcub::detail::ptx_tag<800, no_type>;

void test_ptx_arch_lookup()
{
  using hipcub::detail::type_list;
  using hipcub::detail::ptx_arch_lookup_t;

  {
    using PtxTypes = type_list<TestPolicy350>;
    using Lookup350 = ptx_arch_lookup_t<350, PtxTypes>;
    using Lookup800 = ptx_arch_lookup_t<800, PtxTypes>;
    AssertEquals((std::is_same<Lookup350, TestPolicy350>::value), true);
    AssertEquals((std::is_same<Lookup800, TestPolicy350>::value), true);
  }

  {
    using PtxTypes = type_list<TestPolicy350, TestPolicy600, TestPolicy800>;
    using Lookup350 = ptx_arch_lookup_t<350, PtxTypes>;
    using Lookup520 = ptx_arch_lookup_t<520, PtxTypes>;
    using Lookup600 = ptx_arch_lookup_t<600, PtxTypes>;
    using Lookup700 = ptx_arch_lookup_t<700, PtxTypes>;
    using Lookup800 = ptx_arch_lookup_t<800, PtxTypes>;
    using Lookup860 = ptx_arch_lookup_t<860, PtxTypes>;
    AssertEquals((std::is_same<Lookup350, TestPolicy350>::value), true);
    AssertEquals((std::is_same<Lookup520, TestPolicy350>::value), true);
    AssertEquals((std::is_same<Lookup600, TestPolicy600>::value), true);
    AssertEquals((std::is_same<Lookup700, TestPolicy600>::value), true);
    AssertEquals((std::is_same<Lookup800, TestPolicy800>::value), true);
    AssertEquals((std::is_same<Lookup860, TestPolicy800>::value), true);
  }

}

int main()
{
  test_ptx_arch_lookup();
}
